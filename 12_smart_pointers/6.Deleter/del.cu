
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <algorithm>

struct func // Custom Deleter
{
    void operator()(int *p)
    {
        std::cout<<"\n value freeing = "<<*p;
        free(p);
        std::cout << "\n Pointer freed";
    }
};

void mallocfree(int *p)         // Coustom deletr 2 
{
    free(p);
    std::cout << "\n Coustom deleter 2 ";
}
void normal()
{
    std::unique_ptr<int> p{(int *)malloc(4)};
    *p = 100;
    std::cout << *p << "\n";
    // This can cause problem because every smart pointer will call delete p here
    // But for malloc we need to do free(p)
    // Every smart pointer uses deleter function to call delete
    // so we can have our own implementation of deleter function , so that we can call whatever delete function we want acc to initialization
}

void deleter_example()
{
    std::unique_ptr<int, func> p{(int *)malloc(4), func{}};
    *p = 100;
    std::cout << "\n p = " << *p << "\n";

    std::unique_ptr<int, void (*)(int *)> t{(int *)malloc(4), mallocfree};
    *t = 90;
    std::cout << "\n t = " << *t;
}

void deletr_shared()
{
    std::shared_ptr<int> p{(int *)malloc(4), func{}};           // In shared pointer 1 variable in templete is less to mention
    *p = 100;
    std::cout << "\n p = " << *p << "\n";

    std::shared_ptr<int> t{(int *)malloc(4), mallocfree};
    *t = 90;
    std::cout << "\n t = " << *t;
}

void dynamic()
{
    std::unique_ptr<int> p{new int[5]{1,2,3,4,5}};
    //p[0] = 10;     // cant do it because its not hoe we access dynamic smart pointers
    p.get()[0] = 10; //  ----> now its ok 
    // OR WE CAN DO 
    std::unique_ptr<int[]> k{new int[5]{1,2,3,4,5}};
    k[0]=10;                // --> now its ok 
}
int main()
{
    // Normal issues
    normal();
    // Using delter_function
    deleter_example();
    // Using shared pointer
    deletr_shared();

    //Dynamic arrays
    dynamic();

    return 0;
}