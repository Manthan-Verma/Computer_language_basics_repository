
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// This auto initialization doesn't work in classes
auto fsum(int i, int j)
{
    return i + j;
}
/*
                                // AUTO INITIALIZATION KEYWORD
int main()
{
    auto x{ 5 };                // Using auto for initialization(i.e automatically detects the data type)
    auto y{ 90.78f };           // Detects float data type
    auto sum{ x + y };
    std::cout << "\n sum = " << sum;
    auto func_sum{ fsum(10,20) };
    std::cout << "\n function sum = " << func_sum;
    return 0;
}*/

// Range based for loops

int main()
{
    int a[]{1, 2, 3, 4, 5};
    char p[]{"Hello this is world "};
    for (auto &&i : a)
    {
        std::cout << "\n " << i << "     " << p[i];
    }
    return 0;
}