
#include <hip/hip_runtime.h>
#include <iostream>

// Template code for specific type depending on call is made at compile time . therfore 
// no runtime cost is there
template <typename T>                   // This is how a basic 
auto max(T x, T y)                      // template function is made
{
    return (x > y) ? x : y;
}



// explicit instantiation
template<> auto max(const char *a,const char *b)
{
    std::cout<<"\n type = "<<typeid(a).name();
    return strcmp(a,b) > 0 ? a : b;
}

template<typename T, int m>                 // here m is a non-template parameters so while passing it should be constant
T v(T a)
{
    // even here also we cant do :
    // m++;
    std::cout<<"\n val = "<<m;
    return m;
}

template<typename T, int size>                 // like this it can auto detect size of array
T call_sum(T (&a)[size])
{
    T sum{0};
    for (size_t i = 0; i < size; i++)
    {
        sum += a[i];
    }
    return sum;
}

int main()
{
    // int maximum = max(45,67);
    auto maximum = max<float>(45.7,67);         // Calling according to type 
                                            // calling ----> max<float> is only if to override default type , else
                                            // max() will also work



    // Explicit instantitaton ( if i want to define template function for specific type explicitly)
    
    std::cout<<"\n data = "<<max("A","B");

    std::cout<<"\n maximum is = "<<maximum;

    // non template arguments   
    auto f = v<int,5>(67);          // Here if we put variable in place of 5 it will show error

    float array_1[]{1,78.8,90.5};
    std::cout<<"\n sum = "<<call_sum(array_1);
    return 0;
}