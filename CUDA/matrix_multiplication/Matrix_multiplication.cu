#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <stdlib.h>
#include <stdio.h>
using namespace std;
using namespace std::chrono;
const int N1 = 1000;
const int N2 = 1000;

																								//   GPU ASSIGN_DATA KERNEL

__global__ void assign_data(int* matrix1, int* matrix2)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	//printf("\n threadidx.x = %d , blockidx.x = %d  , blockdim.x = %d  , griddim.x = %d , i = %d", threadIdx.x,blockIdx.x,blockDim.x,gridDim.x,i);
	matrix1[i] = matrix2[i] = i;
}


																							//  GPU DATA MULTIPLICATION EXECUTION SEQUENTIAL 2D


__global__ void multiply_data_seq_2d(int* matrix1, int* matrix2, int* result)
{
	for (int i = 0; i < N1; i++)
	{
		for (int j = 0; j < N1; j++)
		{
			result[i * N1 + j] = 0;
			for (int k = 0; k < N2; k++)
			{
				result[i * N1 + j] += matrix1[i * N2 + k] * matrix2[k * N1 + j];
			}
		}
	}
}

																							
																							//  GPU DATA MULTIPLICATION EXECUTION SEQUENTIAL 1D

__global__ void multiply_data_seq_1d(int* matrix1, int* matrix2, int* result)
{
	for (int i = 0; i < N1*N2; i++)
	{
		result[i] += matrix1[i] * matrix2[i];
	}
}


																							// PARALLEL EXECUTION OF MATRIX MULTIPLICATION ON GPU 2D


__global__ void multiply_parallel(int* matrix1, int* matrix2, int* result)
{
	result[blockIdx.x * gridDim.x + threadIdx.x] = 0;
	for (int k = 0; k < N2; k++)
	{
		result[blockIdx.x * gridDim.x + threadIdx.x] += matrix1[blockIdx.x * blockDim.x + k] * matrix2[k * gridDim.x + threadIdx.x];
	}
}

																								//  CPU COMPUTATION RESULTS 
void matrixsquare()
{
	int hmatrix1[N1][N2], hmatrix2[N2][N1], hmatrixr[N1][N1] = { 0 }, hmatrixf[N1][N1] = { 0 };
	for (int i = 0; i < N1; i++)
	{
		for (int j = 0; j < N2; j++)
		{
			hmatrix1[i][j] = rand() % 10;							// INITIALIZNG THE HMARTIX1
			hmatrix2[j][i] = rand() % 10;							// INITIALIZE THE HMATRIX2
		}
	}
	auto start = high_resolution_clock::now();
	for (int i = 0; i < N1; i++)
	{
		for (int j = 0; j < N1; j++)
		{
			for (int k = 0; k < N2; k++)
			{
				hmatrixr[i][j] += hmatrix1[i][k] * hmatrix2[k][j];
			}
		}
	}
	auto stop = high_resolution_clock::now();
	auto start1 = high_resolution_clock::now();
	for (int i = 0; i < N1; i++)
	{
		for (int k = 0; k < N2; k++)
		{
			for (int j = 0; j < N1; j++)
			{
				hmatrixf[i][j] += hmatrix1[i][k] * hmatrix2[k][j];
			}
		}
	}
	auto stop1 = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start) / 1e3;
	auto duration1 = duration_cast<microseconds>(stop1 - start1) / 1e3;
	cout << "\n Total time taken by the loop for multiplication("<<N1<<") X ("<<N2<<") on cpu with row major approch  is = " << duration.count() << " milliseconds \n";
	cout << "\n Total time taken by the loop for multiplication(" << N1 << ") X (" << N2 << ") on cpu with row coloum approch  is = " << duration1.count() << " milliseconds \n";
}

																									
																								//  GPU COMPUTATION RESULTS 

void GPU_cmputation()
{
	int* matrix1, * matrix2, * result, * result_1d;
	hipMalloc(&matrix1, sizeof(int) * N1 * N2);
	if (hipGetLastError() != hipSuccess)
	{
		cout << "\n Memory allocatio failed inside GPU";
	}
	hipMalloc(&matrix2, sizeof(int) * N2 * N1);
	if (hipGetLastError() != hipSuccess)
	{
		cout << "\n Memory allocatio failed inside GPU";
	}
	hipMalloc(&result, sizeof(int) * N1 * N1);
	if (hipGetLastError() != hipSuccess)
	{
		cout << "\n Memory allocatio failed inside GPU";
	}
	hipMalloc(&result_1d, sizeof(int) * N1 * N2);
	if (hipGetLastError() != hipSuccess)
	{
		cout << "\n Memory allocatio failed inside GPU";
	}
	assign_data << <N1, N2 >> > (matrix1, matrix2);
	hipDeviceSynchronize();
	auto start_seq = high_resolution_clock::now();
	//multiply_data_seq_2d << <1, 1 >> > (matrix1, matrix2, result);
	hipDeviceSynchronize();
	auto stop_seq = high_resolution_clock::now();
	multiply_parallel << <N1, N1 >> > (matrix1, matrix2, result);
	hipDeviceSynchronize();
	auto stop_par = high_resolution_clock::now();
	multiply_data_seq_1d << <1, 1 >> > (matrix1, matrix2, result);
	hipDeviceSynchronize();
	auto stop_extra = high_resolution_clock::now();
	auto duration_seq = duration_cast<microseconds>(stop_seq - start_seq) / 1e3;
	auto duration_par = duration_cast<microseconds>(stop_par - stop_seq) / 1e3;
	auto duration_extra = duration_cast<microseconds>(stop_extra - stop_par) / 1e3;
	cout << "\n Total time taken by the loop for multiplication(" << N1 << ") X (" << N2 << ") on cpu with row major approch  sequential execution on GPU  is = " << duration_seq.count() << " milliseconds \n";
	cout << "\n Total time taken by the loop for multiplication(" << N1 << ") X (" << N2 << ") on cpu with row major approch  parallel execution on GPU is = " << duration_par.count() << " milliseconds \n";
	cout << "\n Total time taken by the loop for multiplication(" << N1 << ") X (" << N2 << ") on cpu with row major approch  parallel execution on GPU is = " << duration_extra.count() << " milliseconds \n";
}
int main()
{
	//cout << "\n Following results are for CPU COMPUTATION \n";
	//matrixsquare();
	cout << "\n \n Following results are for GPU COMPUTATION \n";
	GPU_cmputation();
	return 0;
}