#include "hip/hip_runtime.h"
#include"texture_mem.cuh"

__global__ void t(hipArray * array)
{
    printf("\n data = %f ", tex2D(tex, 0, 0));
    printf("\n data = %f ", tex2D(tex, 0, 1)); // From the output here you will notice that GPU's has coloum major approch rather than row major
    printf("\n data = %f ", tex2D(tex, 1, 0));
    printf("\n data = %f ", tex2D(tex, 1, 1));

    printf("\n cudarray -> %f",array);
}
int main()
{
    
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    float *cuarray;
    float hostarray[2][2]{0, 1, 2, 3};              //  Can make this a pointer array also, both works .
    std::cout << "\n hostarray -> " << hostarray[0][1];
    hipArray *array;

    if (hipMallocArray(&array, &desc, 2, 2) != hipSuccess)
    {
        std::cout << "\n Failed to initalize cuarray";
    }

    if (hipMemcpyToArray(array, 0, 0, hostarray, sizeof(float) * Nx * Ny, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << "\n failed to copy host to gpu ";
    }

    hipBindTextureToArray(tex, array);

    t<<<1, 1>>>(array);
    hipDeviceSynchronize();
    return 0;
}

//    1D USAGE OF TEXTURE MEMORY
/*
__global__ void t()
{
    printf("\n data = %f ",tex1Dfetch(tex,2));
}
int main()
{
    float* cuarray;
    float* hostarray = new float[]{0,1,2,3};
    
    if(hipMalloc(&cuarray,sizeof(float)*Nx*Ny) != hipSuccess)
    {
        std::cout<<"\n Failed to initalize cuarray";
    }

    if(hipMemcpy(cuarray,hostarray,sizeof(float)*Nx*Ny,hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout<<"\n failed to copy host to gpu ";
    }

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture(NULL, tex , cuarray,sizeof(float)*Nx*Ny);

    t<<<1,1>>>();
    hipDeviceSynchronize();

    return 0;
}*/