#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

__global__ void shared()
{
	__shared__ int s;											// THIS VAR IS SHARED ACROSS ALL THREADS IN A THREADBLOCK
	if (threadIdx.x == 0)
		s = 0;
	if (threadIdx.x == 1)
		s += 1;
	if (threadIdx.x == 100)
		s += 2;
	if (threadIdx.x == 0)
		printf("\n S = %d", s);
}

__global__ void shared_barrier()
{
	__shared__ int s;											// THIS VAR IS SHARED ACROSS ALL THREADS IN A THREADBLOCK
	if (threadIdx.x == 0)
	{
		s = 0;
	}
	if (threadIdx.x == 1)
	{
		s += 1;
	}
	__syncthreads();

	if (threadIdx.x == 100)
	{
		s += 2;
	}
	__syncthreads();

	if (threadIdx.x == 0)
	{
		printf("\n S = %d", s);
	}
}

__global__ void classeg(int* matrix,int Threadsize)
{
	__shared__ int a[1024];
	if (threadIdx.x < 1023)
	{
		a[threadIdx.x] = matrix[(blockIdx.x * blockDim.x) + threadIdx.x] + matrix[(blockIdx.x * blockDim.x) + threadIdx.x + 1];
		__syncthreads();
		matrix[(blockIdx.x * blockDim.x) + threadIdx.x] = a[threadIdx.x];
	}
}


/*int main()
{
	/*int Blocksize = 1024, Threadsize = 1024, * matrix, * hmatrix;
	hmatrix = new int[1024 * 1024];
	hipMalloc(&matrix, sizeof(int) * 1024 * 1024);
	for (int i = 0; i < 1024*1024; i++)
	{
		hmatrix[i] = i;
	}
	hipMemcpy(matrix, hmatrix, sizeof(int) * 1024 * 1024, hipMemcpyHostToDevice);
	shared_barrier << <100, Blocksize >> > ();
	hipDeviceSynchronize();
	classeg << <Blocksize, Threadsize >> > (matrix,Threadsize);
	hipDeviceSynchronize();*/
	
	
	/*shared_barrier << <200, 1000 >> > ();
	
	hipDeviceSynchronize();
	return 0;
}*/


//								FOR DYNAMIC SHARED MEMEORY

__global__ void dyshared(int* data)
{
	extern __shared__ int m[];						// shared inside a threadblock 
	m[threadIdx.x] = data[threadIdx.x];
	printf("\n value at %d = %d", threadIdx.x, m[threadIdx.x]);
}

//								 Dynamic Multi-shared
__global__ void dykernel()
{
	extern __shared__ int s[];
	int* p1{ s };
	int* p2{ s + 1 };
	int* p3{ s + 2 };
	if (threadIdx.x == 0)
	{
		s[threadIdx.x] = 0;
		s[threadIdx.x + 1] = 1;
		s[threadIdx.x + 2] = 2;
	}
	else
	{
		printf("\n data = %d  ,  %d ,  %d", *p1, *p2, *p3);
	}

}
int main()
{
	int a;
	std::cout << "enter the no of elements to be shared memeory (is to be <= 1024)";
	std::cin >> a;
	int* data,* data_passed;
	data = new int[1024];

	// Setup for shared memeory 
	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);   // ----> This is used for setting the cach configration for all kernels
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dyshared), hipFuncCachePreferShared);   //-----> This is used for the setting of cache configration for perticular kernel 
	// Here cudaDuncCacheprefershared --> gives all cache to shared 
	//		hipFuncCachePreferEqual		--> divides L1 and shared equally among processes
	//		hipFuncCachePreferNone		---->  Default configration of L1 and shared 
	//		hipFuncCachePreferL1  ---> all cache memory to L1


	for (int i = 0; i < 1024; i++)
	{
		data[i] = i;
	}
	if (hipMalloc(&data_passed, sizeof(int) * 1024) != hipSuccess)
	{
		std::cout << "\nmemeory allocation failed in gpu ";
		return 0;
	}

	if (hipMemcpy(data_passed, data, sizeof(int) * 1024, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cout << "\n memory copy failed from host to device";
		return 0;
	}

	dyshared << <1, a, sizeof(int)* a >> > (data_passed);				// ------> Here third parameter is used for passing the shared memory size to kernel 
	if (hipGetLastError() != hipSuccess)
	{
		std::cout << "\n Kernel launch error ";
		return 0;
	}
	hipDeviceSynchronize();

	dykernel << <1, a, sizeof(int)* a >> > ();				// ------> Here third parameter is used for passing the shared memory size to kernel 
	if (hipGetLastError() != hipSuccess)
	{
		std::cout << "\n Kernel launch error ";
		return 0;
	}
	return 0;
}