#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void dkernel(int N, int* da)
{
	da[threadIdx.x * N] = threadIdx.x;
	printf("\n data[%d] = %d", (threadIdx.x * N), (da[threadIdx.x * N]));
}
int main()
{
	cout << "\n Enter the degree of coalscing(1-32) ";
	int N, * da;
	cin >> N;
	hipMalloc(&da, sizeof(int) * N * 32);
	dkernel << <1, 32 >> > (N,da);
	hipDeviceSynchronize();
	cout << "\n Programe with coalscing " << N << "  created";
	return 0;

}

