#include<iostream>
#include<hip/hip_runtime.h>



__global__ void grading(float* number_device, char* grades)
{
    __shared__ int pass_count,failcount; 
    if(number_device[threadIdx.x]>50)
    {
        grades[threadIdx.x] = 'P';
        atomicAdd(&pass_count,1);
    }
    else
    {
        grades[threadIdx.x] = 'F';
        atomicAdd(&failcount,1);
    }
    __syncthreads();
    if(threadIdx.x ==0)
    {
        printf("\n Therefore on gpu pass count = %d , fail count = %d",pass_count,failcount);
    }

}

int main()
{
    int a,pass_count_cpu{0},fail_count_cpu{0};
    std::cout<<"\n Enter The number of students ";
    std::cin>>a;
    
    float* number = new float[a], * number_device;
    char* grades;
    for (int i = 0; i < a; i++)
    {
        number[i] = rand()%100;
        if(number[i]>50)
        {
            pass_count_cpu++;
        }
        else{
            fail_count_cpu++;
        }

    }
    std::cout<<"\n therefore on cpu passcount = "<<pass_count_cpu<<" , failcount =  "<<fail_count_cpu;
    hipMalloc(&grades,sizeof(char)*a);
    hipMalloc(&number_device,sizeof(float)*a);
    hipMemcpy(number_device,number,sizeof(float)*a,hipMemcpyHostToDevice);

    grading<<<1,a>>>(number_device,grades);

    return 0;
}