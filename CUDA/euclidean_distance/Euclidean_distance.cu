#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <complex>
#include <math.h>
#include <random>
using namespace std;

__global__ void euclidean(float2* ddist, float* check, int total)
{
	float dist = 0, t = 0;
	float2 c;
	c.x = c.y = 0;
	/*for (int i = 0; i < total; i++)
	{
		printf("\n data for thread is %f,  %f ", ddist[threadIdx.x].x, ddist[threadIdx.x].y);
		printf("\n distance data thread %d = %f   %f ", threadIdx.x, ddist[i].x, ddist[i].y);
	}*/
	for (int j = threadIdx.x+1 ; j < total; j++)
	{
		c.x = (ddist[j].x - ddist[threadIdx.x].x)* (ddist[j].x - ddist[threadIdx.x].x);
		c.y = (ddist[j].y - ddist[threadIdx.x].y)* (ddist[j].y - ddist[threadIdx.x].y);
		t = c.x + c.y;
		//printf("\n thread %d distance %f    %f   %f", threadIdx.x, c.x, c.y,t);
		t = sqrt(t);
		if (t > dist)
		{
			dist = t;
		}
	}
	check[threadIdx.x] = dist;
}

__global__ void max(float* check, int total)
{
	float c = 0;
	for (int i = 0; i < total-1; i++)
	{
		printf("\n val = %f", check[i]);
		if (check[i] > c)
		{
			c = check[i];
		}
	}
	printf("\n max distance is %f", c);
}
int main()
{
	const int N = 5;
	float2* hdist, * ddist;
	float* check;
	hdist = new float2[N];
	for (int i = 0; i < N; i++)
	{
		hdist[i].x = rand() % 10;
		hdist[i].y = rand() % 5;
		//cout << "\n Data is " << hdist[i].x <<"  "<< hdist[i].y;
	}
	hipMalloc(&ddist, sizeof(float2) * N);
	hipMalloc(&check, sizeof(float) * (N - 1));
	hipMemcpy(ddist, hdist, sizeof(float2) * N, hipMemcpyHostToDevice);
	euclidean << <1, N-1 >> > (ddist, check, N);
	hipDeviceSynchronize();
	max << <1, 1 >> > (check, N);
	hipDeviceSynchronize();
	return 0;
}
