#include "hip/hip_runtime.h"
﻿#include<iostream>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
__global__ void dod()
{
	int i = threadIdx.x;
	if (i<10)
	{
		printf("\n Hello %d ", i);
		if (i==1)
		{
			printf("\n thread1 ", i);
		}
		else if(i==2)
		{
			printf("\n thread 2 ", i);
		}
	}
	else
		if (i==25)
		{
			printf("\n thread25 %d ", i);
		}
		else
			printf("\n waste %d ", i);
}
int main()
{
	dod << <1, 32 >> > ();
	hipDeviceSynchronize();
	return 0;
}