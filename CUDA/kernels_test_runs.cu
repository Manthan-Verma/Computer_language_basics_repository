#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


#include <complex>
#include <stdio.h>
#include <string>
#include <algorithm>
#include <memory.h>


__constant__ int x_gpu, y_gpu, z_gpu;

constexpr int Nx{512}, Ny{512}, Nz{512};
constexpr int procs{8};

float total_time{0};
hipfftComplex *h_data, *d_data, *d_transpose;

hipEvent_t start, stop;

extern "C++" inline void gpuerrcheck_cudaerror(hipError_t err, int line) // CUFFT ERROR CHECKER
{
    if (err != 0)
    {
        std::cout << "\n cuda error  = " << hipGetErrorString(err) << " , At line " << line;
        exit(0);
    }
}

__global__ void exp()
{
    printf("\n threadidx.x , threadidx.y , threadidx.z = %d , %d , %d", threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void chunk_transpose(hipfftComplex *matrix_data, hipfftComplex *matrix_transpose)
{
    long long int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int d_tmp_z = Nz / procs;
    int Nx_no = i % Nx;
    int Ny_no = (i / Nx) % Ny;
    int Nz_no = i / (Nx * Ny);
    int odd_even = Nz_no / d_tmp_z;
    int put_odd_even = Nz_no % d_tmp_z;
    long long int put_no_slab = (odd_even * Ny * Nx * Nz / procs) + (put_odd_even * Nx) + (Ny_no * Nx * Nz / procs);
    long long int put_no_full = put_no_slab + Nx_no;
    matrix_transpose[put_no_full] = matrix_data[i];
}

__global__ void copy_kernel(hipfftComplex *matrix_data, hipfftComplex *matrix_transpose)
{
    long long int i = threadIdx.x + (blockDim.x * blockIdx.x);
    matrix_transpose[i] = matrix_data[i];
}

__global__ void transpose_slab_type0(hipfftComplex *matrix_data, hipfftComplex *matrix_transpose)
{
    __shared__ hipfftComplex data[256];

    long long int i = threadIdx.x + (blockDim.x * blockIdx.x);
    int z = i % (z_gpu / 2 + 1);
    int y = (i / (z_gpu / 2 + 1)) % Ny;
    int x = (i / ((z_gpu / 2 + 1) * Ny));
    long long int put_no = (x * (z_gpu / 2 + 1)) + (y * (z_gpu / 2 + 1) * Nx) + z;
    
    matrix_transpose[put_no] = matrix_data[i];

    //__syncthreads();
    
    //matrix_transpose[put_no] = data[threadIdx.x];
}
__global__ void transpose_slab_type1(hipfftComplex *matrix_data, hipfftComplex *matrix_transpose)
{
    __shared__ hipfftComplex data[16][16];
    int z = blockIdx.x;
    int y = threadIdx.y + (blockIdx.y * blockDim.y);
    int x = threadIdx.z + (blockDim.z * blockIdx.z);

    long long int input = (x * Ny * (Nz/2+1) ) + (y * (Nz/2+1) )  + z;

    long long int output = (y * (Nz/2+1) * Nx) + (x * (Nz/2+1)) + z;
    

    data[threadIdx.x][threadIdx.y] = matrix_data[input];
    
    __syncthreads();
    
    matrix_transpose[output] = data[threadIdx.x][threadIdx.y];
}
__global__ void transpose_slab_type2(hipfftComplex *matrix_data, hipfftComplex *matrix_transpose)
{
    __shared__ hipfftComplex data[16][16];
    int z = blockIdx.z;
    int y = threadIdx.y + (blockIdx.y * blockDim.y);
    int x = threadIdx.x + (blockDim.x * blockIdx.x);

    long long int input = (x * Ny * (Nz/2+1)) + (y * (Nz/2+1)) + z;

    long long int output = (y * (Nz/2+1) * Nx) + (x * (Nz/2+1)) + z;
    

    data[threadIdx.x][threadIdx.y] = matrix_data[input];
    
    __syncthreads();
    
    matrix_transpose[output] = data[threadIdx.x][threadIdx.y];
}

/*__global__ void transpose_slab_coalsced(cufftComplex *matrix_data, cufftComplex *matrix_transpose, long long int Ny, long long int Nx)
{
    __shared__ cufftComplex data[16][16];
    long long int i = (threadIdx.x * 32) + (blockDim.x * 32 * blockIdx.x);
    for (long long int loop = 0; loop < 32; loop++)
    {
        i += loop;
        int z = i % (z_gpu / 2 + 1);
        int y = (i / (z_gpu / 2 + 1)) % Ny;
        int x = (i / ((z_gpu / 2 + 1) * Ny));
        long long int put_no = (x * (z_gpu / 2 + 1)) + (y * (z_gpu / 2 + 1) * Nx) + z;
        matrix_transpose[put_no] = matrix_data[i];
    }
}*/

void init_var()
{
    h_data = (hipfftComplex *)malloc(sizeof(hipfftComplex) * Nx * Ny * Nz);
    for (int i = 0; i < Nx; i++)
    {
        //std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            //std::cout << "\n";
            for (int k = 0; k < Nz; k++)
            {
                h_data[(i * Ny * Nz) + (j * Nz) + k].x = rand() % 10;
                h_data[(i * Ny * Nz) + (j * Nz) + k].y = rand() % 10;
                //std::cout << "  " << data[(i * Ny * Nz) + (j * Nz) + k].x << "," << data[(i * Ny * Nz) + (j * Nz) + k].y; //<< "(" << i << "," << j << "," << k << ")";
            }
        }
    }

    hipMalloc(&d_data, sizeof(hipfftComplex) * Nx * Ny * Nz);
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1);

    hipMalloc(&d_transpose, sizeof(hipfftComplex) * Nx * Ny * Nz);
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1);
}
int main()
{
    hipEventCreate(&start);
    hipEventCreate(&stop);

    init_var();

    dim3 grid_chunk{(Nz/2+1)*Ny*Nx/256,1,1};
    dim3 block_chunk{256,1,1};

    dim3 grid_copy{(Nz/2+1)*Ny*Nx/256,1,1};
    dim3 block_copy{256,1,1};

    dim3 grid_slab_type0{(Nz/2+1)*Ny*Nx/256,1,1};
    dim3 block_slab_type0{256,1,1};

    dim3 grid_slab_type1((Nz/2+1),Ny/16,Nx/16);
    dim3 block_slab_type1(1,16,16);

    dim3 grid_slab_type2(Nx / 16 , Ny / 16, (Nz / 2 + 1));
    dim3 block_slab_type2(16, 16, 1);
    
    //dim3 grid_slab_c((Nz / 2 + 1) * Ny * Nx / (256 * 32), 1, 1);
    //dim3 block_slab_c(256, 1, 1);

    hipMemcpyToSymbol(HIP_SYMBOL(x_gpu), &Nx, sizeof(int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1);
    hipMemcpyToSymbol(HIP_SYMBOL(y_gpu), &Ny, sizeof(int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1);
    hipMemcpyToSymbol(HIP_SYMBOL(z_gpu), &Nz, sizeof(int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1);

    // warmp up
    transpose_slab_type1<<<grid_slab_type1, block_slab_type1>>>(d_data, d_transpose);
    transpose_slab_type2<<<grid_slab_type2, block_slab_type2>>>(d_data, d_transpose);
    

    // Actual computation

    hipEventRecord(start);
    for (int i = 0; i < 100; i++)
    {
        copy_kernel<<<grid_copy, block_copy>>>(d_data, d_transpose);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);
    std::cout << "\n total time copy kernel = " << total_time / 100 << " ms";

    hipEventRecord(start);
    for (int i = 0; i < 100; i++)
    {
        transpose_slab_type0<<<grid_slab_type0, block_slab_type0>>>(d_data, d_transpose);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);
    std::cout << "\n total time type0 = " << total_time / 100 << " ms";
    
    /*cudaEventRecord(start);
    for (int i = 0; i < 100; i++)
    {
        transpose_slab_type1<<<grid_slab_type1, block_slab_type1>>>(d_data, d_transpose);
    }
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&total_time, start, stop);
    std::cout << "\n total time type1 = " << total_time / 100 << " ms";

    cudaEventRecord(start);
    for (int i = 0; i < 100; i++)
    {
        transpose_slab_type2<<<grid_slab_type2, block_slab_type2>>>(d_data, d_transpose);
    }
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&total_time, start, stop);
    std::cout << "\n total time type2  = " << total_time / 100 << " ms";*/

    hipEventRecord(start);
    for (int i = 0; i < 100; i++)
    {
        chunk_transpose<<<grid_chunk, block_chunk>>>(d_data, d_transpose);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);
    std::cout << "\n total time chunk = " << total_time / 100 << " ms";

    return 0;
}