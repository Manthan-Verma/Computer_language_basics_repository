#include "hip/hip_runtime.h"
#include "matrix_trans.cuh"

__global__ void trans2_cut_with_y(float2 *matrix_data, float2 *matrix_transpose)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int no = i / d;
    int k = i - (no * d);
    int j = ((k % (Nx)) * (Ny/procs)) + (k / (Nx));
    int l = (no * d) + j;
    matrix_transpose[l] = matrix_data[i];
}

__global__ void trans2_cut_with_x(float2 *matrix_data, float2 *matrix_transpose)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int Nx_no = i % Nx;
    int Ny_no = (i/Nx)%Ny;
    int Nz_no = i / (Nx*Ny);
    int odd_even = Nx_no/d_tmp;
    int put_odd_even = Nx_no%d_tmp;
    int put_no_slab = (put_odd_even*Ny*procs) + (Ny*odd_even) + Ny_no ;
    int put_no_full = put_no_slab + (Nz_no*(Nx*Ny));
    matrix_transpose[put_no_full] = matrix_data[i];
}

__global__ void trans2_cut_with_z_allocated_along_x(float2 *matrix_data, float2 *matrix_transpose)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int Nx_no = i % Nx;
    int Ny_no = (i/Nx)%Ny;
    int Nz_no = i / (Nx*Ny);
    int Real_Nz_no = Nz - Nz_no-1;
    int odd_even = Nz_no/d_tmp_z;
    int put_odd_even = Nz_no % d_tmp_z;
    int put_no_slab = (odd_even*Ny*Nx*Nz/procs) + (put_odd_even*Nx) + (Ny_no*Nx*Nz/procs);
    int put_no_full = put_no_slab + Nx_no;
    matrix_transpose[put_no_full] = matrix_data[i];
}

__global__ void trans2_cut_with_z_reverse_allocated_along_x(float2 *matrix_data, float2 *matrix_transpose)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    int Nx_no = i % Nx;
    int Ny_no = (i/Nx)%(Nz/procs);
    int Nz_no = i / (Nx*(Nz/procs));
    //int Real_Nz_no = Nz - Nz_no-1;
    int odd_even = Nz_no/Ny;
    int put_odd_even = (Nz_no % Ny);
    int put_no_slab = (odd_even*Ny*Nx*Nz/procs) + (put_odd_even*Nx) + (Ny_no*Nx*Ny);
    int put_no_full = put_no_slab + Nx_no;
    matrix_transpose[put_no_full] = matrix_data[i];
}
__global__ void transpose(float2 *matrix_data, float2 *matrix_transpose)
{
    long long int i = threadIdx.y + (blockDim.y * threadIdx.x) + (blockIdx.x * blockDim.x * blockDim.y);
    long long int j = (threadIdx.y * blockDim.x) + threadIdx.x + (blockIdx.x * blockDim.x * blockDim.y);
    matrix_transpose[j].x = matrix_data[i].x;
    matrix_transpose[j].y = matrix_data[i].y;
}

__global__ void transpose_slab(float2 *matrix_data, float2 *matrix_transpose)
{
    int i = threadIdx.x + (blockDim.x*blockIdx.x) + (blockIdx.y*gridDim.x*blockDim.x);
    int j = ((threadIdx.x*gridDim.x)+blockIdx.x)+(blockIdx.y*blockDim.x*gridDim.x);
    matrix_transpose[i]=matrix_data[j];
}
__global__ void transpose_slab_in_z_y_dir(float2 *matrix_data, float2* matrix_transpose)
{
    long long int i = threadIdx.x + (blockDim.x * blockIdx.x);
    int z = i % Nz;
    int y = (i / Nz) % Ny;
    int x = (i / (Nz*Ny));
    long long int put_no = (x*Nz) + (y * Nz * Nx) + z;
    matrix_transpose[put_no] = matrix_data[i];
}
void test2(float2* data)
{
    float2* transpose,* check;
    check = (float2*)calloc(Nx*Ny*Nz,sizeof(float2));
    hipMalloc(&transpose,sizeof(float2)*Nx*Ny*Nz);
    hipDeviceSynchronize();
    //dim3 block(Ny,1,1);
    //dim3 grid(Nx,Nz,1);
    //transpose_slab<<<grid,block>>>(data,transpose);
    transpose_slab_in_z_y_dir<<<1,(Nx*Ny*Nz)>>>(data,transpose);

    hipMemcpy(check,transpose,sizeof(float2)*Nx*Ny*Nz,hipMemcpyDeviceToHost);
    std::cout << "\n\n data we got after transpose = ";
    for (int i = 0; i < Ny; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Nx; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nz; k++)
            {
                std::cout << "  " << check[(i * Nx * Nz) + (j * Nz) + k].x << "," << check[(i * Nx * Nz) + (j * Nz) + k].y;
            }
        }
    }

}
void Matrix_transpose_gpu(float2 *data)
{
    //dim3 block{Ny, Nz, 1};
    //dim3 grid{Nx, 1, 1};
    float2 *matrix_transpose_gpu, *matrix_transpose_gpu_check;
    matrix_transpose_gpu_check = (float2 *)malloc(sizeof(float2) * Nx * Nz * Ny);

    if (hipMalloc(&matrix_transpose_gpu, sizeof(float2) * Nx * Ny * Nz) != hipSuccess)
    {
        std::cout << "\n Failed to allocate memory on gpu for transpose ";
        return;
    }

    trans2_cut_with_z_allocated_along_x<<<1, (Nx * Ny * Nz)>>>(data, matrix_transpose_gpu);

    trans2_cut_with_z_reverse_allocated_along_x<<<1, (Nx * Ny * Nz)>>>(matrix_transpose_gpu, data);

    if (hipMemcpy(matrix_transpose_gpu_check, data, sizeof(float2) * Nx * Nz * Ny, hipMemcpyDeviceToHost) != hipSuccess)
    {
        std::cout << "\n failed to copy data from device to host ";
        return;
    }

    std::cout << "\n \n Matrix Transpose gpu :: ";
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                std::cout << "  " << matrix_transpose_gpu_check[(i * Ny * Nx) + (j * Nx) + k].x << "," << matrix_transpose_gpu_check[(i * Ny * Nx) + (j * Nx) + k].y;
            }
        }
    }
}

void Matrix_transpose_cpu(float2 *data)
{
    float2 *matrix_transpose_cpu;
    matrix_transpose_cpu = (float2 *)malloc(sizeof(float2) * Nx * Ny * Nz);

    std::cout << "\n Matrix Transpose  CPU :: ";
    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        long long int no = i / d;
        long long int k = i - (no * d);
        long long int j = ((k % Nx) * (Ny / procs)) + (k / Nx);
        long long int l = (no * d) + j;
        matrix_transpose_cpu[l] = data[i];
    }
    
    for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int no_slab1 = i / ((Ny/procs) * (Nx*procs));
        int no_pencil = i % (Nx*procs);
        int h = i / (Nx*procs);
        h %= (Ny/procs);
        int j = (no_slab1 * (Ny/procs) * (Nx*procs)) + (no_pencil * (Ny/procs)) + h;
        data[i] = matrix_transpose_cpu[j];
    }
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < (Ny / procs); j++)
        {
            std::cout << "\n";
            for (int k = 0; k < (Nx*procs); k++)
            {
                std::cout << "  " << data[(i * (Ny / procs) * (Nx * procs)) + (j * (Nx*procs)) + k].x << "," << data[(i * (Nx * procs) * (Ny / procs)) + (j * (Nx*procs)) + k].y;
            }
        }
    }
    reverse_testing_slab(data);
}

void matrix_dim_change_cpu(float2 *data)
{
    float2 *matrix_dim_change_cpu;
    matrix_dim_change_cpu = (float2 *)malloc(sizeof(float2) * Nx * Ny * Nz);
    // rotating a matrix from x*y*z to anti-clockwise wrt y axis 90degree
    std::cout << "\n Matrix DIM CHANGE CPU :: ";
    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        int j{(i % Nz)};
        int k{i / (Nz)};
        k %= Ny;
        int h{i / (Nz * Ny)};
        long long int m{(j * Nx * Ny) + (k * Nx) + h};
        //std::cout << "\n j = " << j << " , k = " << k << " , h = " << h << "        ,m = " << m;

        matrix_dim_change_cpu[i] = data[m];
    }
    for (int i = 0; i < Nx; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nz; k++)
            {
                std::cout << "   " << matrix_dim_change_cpu[(i * Ny * Nz) + (j * Nz) + k].x << "," << matrix_dim_change_cpu[(i * Ny * Nz) + (j * Nz) + k].y;
            }
        }
    }
    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        int j{(i % Nx)};
        int k{i / (Nx)};
        k %= Ny;
        int h{i / (Nx * Ny)};
        long long int m{(j * Nz * Ny) + (k * Nz) + h};
        //std::cout << "\n j = " << j << " , k = " << k << " , h = " << h << "        ,m = " << m;

        data[i] = matrix_dim_change_cpu[m];
    }
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                std::cout << "   " << data[(i * Ny * Nx) + (j * Nx) + k].x << "," << data[(i * Ny * Nx) + (j * Nx) + k].y;
            }
        }
    }
}

void reverse_testing_slab(float2* data)
{
    float2 *transpose = (float2*)calloc(Nx * Ny * Nz,sizeof(float2));
    for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int no_slab1 = i / ((Nx*procs) * (Ny/procs));
        int no_pencil = i % (Ny/procs);
        int h = i / (Ny/procs);
        h %= (Nx*procs);
        int j = (no_slab1 * (Nx*procs) * (Ny/procs)) + (no_pencil * (Nx*procs)) + h;
        transpose[i] = data[j];
    }
    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        int no = i / d;
        int k = i - (no * d);
        int j = ((k % (Ny / procs)) * Nx) + (k / (Ny / procs));
        int l = (no * d) + j;
        data[l] = transpose[i];
    }
    std::cout << "\n matrix initiated by reversing it back = : ";
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                std::cout << "  " << data[(i * Ny * Nx) + (j * Nx) + k].x << "," << data[(i * Ny * Nx) + (j * Nx) + k].y; //<< "(" << i << "," << j << "," << k << ")";
            }
        }
    }
}

void matrix_transpose_cpu_slab(float2 *data)
{
    float2 *transpose = (float2*)calloc(Nx * Ny * Nz,sizeof(float2));

    for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int no_slab1 = i / (Nx * Ny);
        int no_pencil = i % Ny;
        int h = i / Ny;
        h %= Nx;
        int j = (no_slab1 * Nx * Ny) + (no_pencil * Nx) + h;
        transpose[i] = data[j];
    }
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Nx; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Ny; k++)
            {
                std::cout << "   " << transpose[(i * Ny * Nx) + (j * Ny) + k].x << "," << transpose[(i * Ny * Nx) + (j * Ny) + k].y;
            }
        }
    }
    for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int no_slab1 = i / (Nx * Ny);
        int no_pencil = i % Nx;
        int h = i / Nx;
        h %= Ny;
        int j = (no_slab1 * Nx * Ny) + (no_pencil * Ny) + h;
        data[i] = transpose[j];
    }
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                std::cout << "   " << data[(i * Ny * Nx) + (j * Nx) + k].x << "," << data[(i * Ny * Nx) + (j * Nx) + k].y;
            }
        }
    }
}

void test(float2 *data)
{
    float2 *matrix_dim_change_cpu;
    matrix_dim_change_cpu = (float2 *)malloc(sizeof(float2) * Nx * Ny * Nz);
    // rotating a matrix from x*y*z to anti-clockwise wrt y axis 90degree
    std::cout << "\n Matrix DIM CHANGE CPU :: ";
    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        int j{(i % Nz)};
        int k{i / (Nz)};
        k %= Ny;
        int h{i / (Nz * Ny)};
        long long int m{(j * Nx * Ny) + (k * Nx) + h};
        //std::cout << "\n j = " << j << " , k = " << k << " , h = " << h << "        ,m = " << m;

        matrix_dim_change_cpu[i] = data[m];
    }

    for (int i = 0; i < Nx; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nz; k++)
            {
                std::cout << "   " << matrix_dim_change_cpu[(i * Ny * Nz) + (j * Nz) + k].x << "," << matrix_dim_change_cpu[(i * Ny * Nz) + (j * Nz) + k].y;
            }
        }
    }
    for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int no_slab1 = i / (Nz * Ny);
        int no_pencil = i % Ny;
        int h = i / Ny;
        h %= Nz;
        int j = (no_slab1 * Nz * Ny) + (no_pencil * Nz) + h;
        data[i] = matrix_dim_change_cpu[j];
    }

    for (int i = 0; i < Nx; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Nz; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Ny; k++)
            {
                std::cout << "   " << data[(i * Ny * Nz) + (j * Ny) + k].x << "," << data[(i * Ny * Nz) + (j * Ny) + k].y;
            }
        }
    }
    for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int no_slab1 = i / (Nz * Ny);
        int no_pencil = i % Nz;
        int h = i / Nz;
        h %= Ny;
        int j = (no_slab1 * Nz * Ny) + (no_pencil * Ny) + h;
        matrix_dim_change_cpu[i] = data[j];
    }
    for (int i = 0; i < Nx; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nz; k++)
            {
                std::cout << "   " << matrix_dim_change_cpu[(i * Ny * Nz) + (j * Nz) + k].x << "," << matrix_dim_change_cpu[(i * Ny * Nz) + (j * Nz) + k].y;
            }
        }
    }
}

void transpose_wrt_x(float2 *data)
{
    float2 *transpose;
    transpose = (float2 *)malloc(sizeof(float2) * Nx * Ny * Nz);
    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        int k = (i / Nx) % Nz;
        int l = i % Nx;
        int h = i / (Nx * Nz);
        int j = (Nx * Ny * (Nz - k - 1)) + (h * Nx) + l;
        transpose[i] = data[j];
    }

    for (int i = 0; i < Ny; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Nz; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                std::cout << "   " << transpose[(i * Nx * Nz) + (j * Nx) + k].x << "," << transpose[(i * Nx * Nz) + (j * Nx) + k].y;
            }
        }
    }
}

void transpose_wrt_x_3d(float2 *data)
{
    float2 transpose[Ny][Nz][Nx];
    //float2 transpose[Ny][Ny][Nx]
    for (int i = 0; i < Nz; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                transpose[j][i][k] = data[(i * Nx * Ny) + (j * Nx) + k];
                std::cout << "  " << transpose[j][i][k].x << "," << transpose[j][i][k].y;
            }
        }
    }
    std::cout << " \n\n\n " << transpose[0][2][1].x << "," << transpose[0][2][1].y;
}

void transpose_y_z_keeping_x_fixed(float2 *a)
{
    // m is dimension of fast axis , p is dimension of slow axis
    float2 *b,*c;
    hipMalloc(&b, sizeof(float2) * Nx * Ny * Nz);
    c = (float2 *)malloc(sizeof(float2) * Nx * Ny * Nz);
    /*for (int i = 0; i < (Nx * Ny * Nz); i++)
    {
        int coloum = i % Nx;
        int multiply_no = i / Nx;
        int row = multiply_no % Ny;
        int slab_front = i / (Nx * Ny);
        int j = (row * Nx * Nz) + (slab_front * Nx) + coloum;
        b[j] = a[i];
    }*/

    transpose_y_z_with_x_fixed<<<1, Total_data_size>>>(a, b);

    hipMemcpy(c,b,sizeof(float2)*Nx*Ny*Nz,hipMemcpyDeviceToHost);

    std::cout << "\n data we got after transpose = "
              << "\n";
    for (int i = 0; i < Ny; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Nz; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nx; k++)
            {
                std::cout << "  " << c[(i * Nx * Nz) + (j * Nx) + k].x << "," << c[(i * Nx * Nz) + (j * Nx) + k].y;
            }
        }
    }
}

void run(float2* data)
{
    float2* transpose;
    transpose= (float2*)calloc((Nx*Ny*Nz),sizeof(float2));

    for (int i = 0; i < Nx * Ny * Nz; i++)
    {
        int no = i / d;
        int k = i - (no * d);
        int j = ((k % (Ny / procs)) * Nx) + (k / (Ny / procs));
        int l = (no * d) + j;
        data[l] = transpose[i];
    }
}
int main()
{
    float2 *data, *matrix;
    data = (float2 *)malloc(sizeof(float2) * Nx * Ny * Nz);

    std::cout << "\n matrix initiated = : ";
    for (int i = 0; i < Nx; i++)
    {
        std::cout << "\n\n";
        for (int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (int k = 0; k < Nz; k++)
            {
                data[(i * Ny * Nz) + (j * Nz) + k].x = rand() % 10;
                data[(i * Ny * Nz) + (j * Nz) + k].y = rand() % 10;
                std::cout << "  " << data[(i * Ny * Nz) + (j * Nz) + k].x << "," << data[(i * Ny * Nz) + (j * Nz) + k].y; //<< "(" << i << "," << j << "," << k << ")";
            }
        }
    }

    if (hipMalloc(&matrix, sizeof(float2) * Nx * Ny * Nz) != hipSuccess)
    {
        std::cout << "\n Failed to allocate memory on gpu  for normal data ";
        return 1;
    }

    if (hipMemcpy(matrix, data, sizeof(float2) * Nx * Ny * Nz, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << "\n failed to copy data from host to device ";
        return 1;
    }

    test2(matrix);
    //test(data);
   // matrix_transpose_cpu_slab(data);
    //transpose_y_z_keeping_x_fixed(matrix);
    //transpose_wrt_x(data);
    //transpose_wrt_x_3d(data);
    //matrix_dim_change_cpu(data);
    //Matrix_transpose_cpu(data);
    //Matrix_transpose_gpu(matrix);
    return 0;
}
