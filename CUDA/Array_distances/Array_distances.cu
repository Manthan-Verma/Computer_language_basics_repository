#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<iostream>
#include<complex>
#include<algorithm>
using namespace std;

__global__
void cal(complex<float>* dcoordinates, int N, float* max)
{
	int i = threadIdx.x;
	complex<float> mag=0;
	for (int j = i; j < N; j++)
	{
		complex<float> k = dcoordinates[i] - dcoordinates[j];
		if (norm(k) > norm(mag))
		{
			mag = k;
		}
	}
	max[i] = norm(mag);
}
int main()
{
	int N;
	cout << "\n Enter the no of positions to cal coordinates for ";
	cin >> N;
	complex<float>* hcoordinates, * dcoordinates;
	float* max, * hmax;
	hmax = new float[N - 1];											// Max on host 
	hipMalloc(&max, sizeof(float) * (N - 1));								// Assigning value to device 
	hipMalloc(&dcoordinates, sizeof(complex<float>) * N);					//  DEVICE MEMORY ALLOCATION
	hcoordinates = new complex<float>[N];									// HOST MEMORY ALLOCATION
	for (int i = 0; i < N; i++)
	{
		cout << "\n Enter the " << i << "th coordinates x and then y \n";
		cin >> hcoordinates[i];
		cout << "\n Entered value is " << hcoordinates[i];
	}
	hipMemcpy(dcoordinates, hcoordinates, sizeof(complex<float>) * N, hipMemcpyHostToDevice);   // Memory copy from Host to Device
	cal << <1, (N - 1) >> > (dcoordinates, N, max);
	hipMemcpy(hmax, max, sizeof(float) * (N - 1), hipMemcpyDeviceToHost);
	float* maximum = max_element(hmax, hmax + N - 1);
	cout << "\n Max distance is " << maximum[0];
	return 0;

}

