#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <random>
#include <math.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__ void dkernel(float* darr, int k)
{
	float c = 0;
	for (int i = k*threadIdx.x ; i < k*(threadIdx.x+1); i++)
	{
		if (darr[i] > c)
		{
			c = darr[i];
		}
	}
	darr[threadIdx.x * k] = c;
	printf("\n thread is %d , c = %f", threadIdx.x, darr[threadIdx.x * k]);
}
int main()
{
	const int N = 10000;
	int k = 1000,block;
	float* harr, * darr;
	harr = new float[N];
	hipMalloc(&darr, sizeof(float) * N);
	//hipMalloc(&data, sizeof(float) * (N / k));
	for (int i = 0; i < N; i++)
	{
		harr[i] = rand() % 20;
		cout << "\n data is " << harr[i];
	}
	block = N / k;
	hipMemcpy(darr, harr, sizeof(float) * N, hipMemcpyHostToDevice);
	dkernel << <1, N / k >> > (darr, k);
	hipDeviceSynchronize();
	dkernel << <1, 1 >> > (darr, N);
	return 0;
}