﻿#include <iostream>
#include <hip/hip_runtime.h>



// Using constexpr
/*constexpr int m{ 99 };                                              // ---> availaible in the main memory so its availaible to both gpu and cpu 

__global__ void show()
{
    printf("\n data is = %d ", m);
}

int main()
{
    show << <1, 1 >> > ();
    cudaDeviceSynchronize();

    return 0;

}*/


// For Normal data 
__constant__ int m;
__global__ void show()
{
    printf("\n data is = %d " , m);
}


int main()
{
    int a{10};
    hipMemcpyToSymbol( HIP_SYMBOL(m), &a , sizeof(int));
    show<<<1,1>>>();
    hipDeviceSynchronize();
    a = 90;
    hipMemcpyToSymbol( HIP_SYMBOL(m), &a , sizeof(int));
    show<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;

}



// For Array 
/*
__constant__ int m[10];

__global__ void show()
{
    printf("\n data is = %d " , m[2]);
}

int main()
{
    int* a = new int[10]{0,1,2,3,4,5,6,7,8,9};
    cudaMemcpyToSymbol( m, a , sizeof(int)*10);
    show<<<1,1>>>();
    cudaDeviceSynchronize();

    return 0;

}*/