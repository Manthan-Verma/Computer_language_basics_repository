#include<iostream>
#include<hip/hip_runtime.h>



__global__ void k(int* p)
{
    *p =0;
    printf("\n *p = %d",*p);
}

int main()
{
    int * x, * y;
    hipMalloc(&x,sizeof(int));
    k<<<2,10>>>(x);
    hipDeviceSynchronize();
    y = x;
    hipFree(y);
    k<<<2,10>>>(y);
    hipDeviceSynchronize();
    return 0;
}