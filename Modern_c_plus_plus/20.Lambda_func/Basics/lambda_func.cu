
#include <hip/hip_runtime.h>
#include <iostream>
// #include <memory>

using comparator = bool (*)(int, int);

template <typename T, int size>
void sort(T (&arr)[size], comparator comp)
{
    for (int i = 0; i < size - 1; i++)
    {
        for (int j = 0; j < size - 1; j++)
        {
            if (comp(arr[j], arr[j + 1]))  // what is we want to sort strings (we write whole algorithm for just this one statement??)
            {                              // We must find a better way . Function pointer can be a way . But this is not a good way
                T tmp = std::move(arr[j]); // As it cannot be inlined so it will cost time. therefre we will use function objects instead of pointers
                arr[j] = std::move(arr[j + 1]);
                arr[j + 1] = std::move(tmp);
            }
        }
    }
}

template <typename T, int size, typename comparator>
void sort1(T (&arr)[size], comparator comp)
{
    for (int i = 0; i < size - 1; i++)
    {
        for (int j = 0; j < size - 1; j++)
        {
            if (comp(arr[j], arr[j + 1]))  // what is we want to sort strings (we write whole algorithm for just this one statement??)
            {                              // We must find a better way . Function pointer can be a way . But this is not a good way
                T tmp = std::move(arr[j]); // As it cannot be inlined so it will cost time. therefre we will use function objects instead of pointers
                arr[j] = std::move(arr[j + 1]);
                arr[j + 1] = std::move(tmp);
            }
        }
    }
}

bool Comp(int x, int y)
{
    return x > y;
}

bool Comp1(int x, int y)
{
    return x < y;
}

struct comp2
{
    bool operator()(int x, int y)
    {
        return x > y;
    }

};

int main()
{
    int arr[]{3, 4, 78, 90, 100, 4, 1, 2};
    // int *arr = (int*)malloc(10*sizeof(int));
    // arr[0] = 0;
    // arr[1] = 12;
    // arr[2] = 11;
    // arr[3] = 7;
    // arr[4] = 90;
    // arr[5] = 87;
    // arr[6] = 8;
    // arr[7] = 6;
    // arr[8] = 1;
    // arr[9] = 9;

    sort(arr, Comp1);

    for (auto &&i : arr)
    {
        std::cout << "\n arr = " << i;
    }
    // Here is i do 
    comp2 co;
    sort1(arr,co);              // This way call to struct comp will be inlined and performs better 
    return 0;
}