#include <iostream>
#include <stdlib.h>
#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>

                                        // User defined literals should start with _ always , literals without _ are in c++ stl library preddefined.
                                        // Only 4 types allowed ---> long double , unsinged long long , const char* , char ;
                                     
class Distance
{
private:
    long double D_kilometers;

public:
    Distance(long double m) : D_kilometers{m}
    {
    }
    long double get_km() const
    {
        return D_kilometers;
    }
    void set_km(long double val)
    {
        D_kilometers = val;
    }
    ~Distance() = default;
};

Distance operator"" _mi(long double val)                // Syntax for making literals 
{
    return {val * 1.6};
}

long double operator"" _meters(long double val)
{
    return {val / 1000};
}
int main()
{
    Distance dist{32};       // Normal invokation
    Distance dist1{45.0_mi}; // Using literals invokation
    std::cout << "\n " << dist.get_km();
    std::cout << "\n " << dist1.get_km();
    long double a{1500.0_meters};
    std::cout << "\n total = " << a << " km";
    return 0;
}