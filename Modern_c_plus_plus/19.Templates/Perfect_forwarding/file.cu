
#include <hip/hip_runtime.h>
#include<iostream>

template<typename T>
void call(const T &&a,const T &&b)
{
    // here lets say i call a object making of a class
    // say --> class_name obj(a,b). --> therefore here 
    // actually a and b is not passed as temprary , but as L values . therfore to 
    // enable passing as it is we do --> class_name obj(std::forward(a),std::forward(b))
}

int main()
{
    call(56,78);
    return 0;
}