
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <memory>
#include <algorithm>
#include <string>
#include <sstream>
#include <initializer_list>
#include <cassert>
                                        /* Pourpose of initializer list is to define object just like an array , espacially if its a container class
                                           1 . Represents an array of objects.
                                           2. Constructs automatically from a braced list of elements --> { }
                                           3. Acess to elements through iterators (like range based for loop)*/     
class Bag
{
private:
    int arr[10];
    int size{};

public:
    Bag(std::initializer_list<int> elements)
    {
        assert(elements.size() < 10);
        for (auto &&i : elements)
        {
            add(i);
        }
    }
    void add(int val)
    {
        assert(size < 10);
        arr[size++] = val;
    }
    void remove()
    {
        --size;
    }
    int operator[](int index)
    {
        return arr[index];
    }
    int getsize() const
    {
        return size;
    }
    ~Bag() = default;
};
int main()
{
    std::string name{"Manthan Verma"};
    int arr[5]{1, 2, 3, 4, 5};
    std::initializer_list<int> array{2, 2, 3, 4, 5};
    for (auto &&i : array)
    {
        std::cout << "\n val = " << i;
    }
    std::cout<<"\n array = "<<array.begin()[3];
    Bag n{1,2,3,4,5};
    std::cout<<"\n size initialized = "<<n.getsize();
}
