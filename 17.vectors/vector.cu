#include "hip/hip_runtime.h"
#include"vector.cuh"
                            /* This vector is a class templete 
                             1.For dynamic memory allocation for pointers at runtime we use vectors 
                             2 .  */



void vector_basics()
{
    std::vector<int> val{1,2,3,4,5};            // Initialization
    std::vector<float2> comp;
    
    // Printing or Accesing elements
    for (auto &&i : val)
    {
        std::cout<<"\n value  = "<<i;
    }

    std::cout<<"\n value 1= "<<val[0];
    val[2] = 100;
    for (int i = 0; i < val.size(); i++)            // Another way of accesing 
    {
        std::cout<<"\n val = "<<val[i];
    }
    
    auto it = val.begin();          // return begning address
    std::cout<<"\n value at 3 = "<<*it<<" , at next = "<<*(++it);

    // Deleting and chnging
    std::cout<<"\n "; 
    val.erase(it);
    for (auto &&i : val)
    {
        std::cout<<i<<" ";
    }
    
    // INSERTION
    std::cout<<"\n";
    val.insert(it,2);           // insert(pos, value);
    for (auto &&i : val)
    {
        std::cout<<i<<" ";
    }
    
    val.push_back(90); //--> added at the end of the vector
}

int main()
{
    vector_basics();
    std::cout<<"\n time = "<<__TIME__;                  // This macro gives current time 
    return 0;
}